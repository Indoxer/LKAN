#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// TODO: Optimize kernels memory access

template <typename scalar_t>
__global__ void fftkan_cuda_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t, 2> X,
    const torch::PackedTensorAccessor32<scalar_t, 2> W,
    const torch::PackedTensorAccessor32<scalar_t, 2> S,
    const torch::PackedTensorAccessor32<scalar_t, 4> C,
    torch::PackedTensorAccessor32<scalar_t, 2> Y,
    const int B, const int I, const int O, const int G)
{
    // X [B, I]
    // W [O, I]
    // S [O, I]
    // C [O, I, 2, G]
    // -> Y [B, O]
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int o = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < B && o < O)
    {
        scalar_t sum = 0.0f;
        for (int i = 0; i < I; i++)
        {
            scalar_t g_sum = 0.0f;
            for (int g = 0; g < G; g++)
            {
                scalar_t v_sin, v_cos;
                sincos((g + 1) * X[b][i], &v_sin, &v_cos);
                g_sum += C[o][i][0][g] * v_cos;
                g_sum += C[o][i][1][g] * v_sin;
            }
            sum += S[o][i] * g_sum;
            sum += W[o][i] * X[b][i] / (1.0f + expf(-X[b][i]));
        }

        Y[b][o] = sum;
    }
}

template <typename scalar_t>
__global__ void fftkan_cuda_backward_kernel_WSC(
    const torch::PackedTensorAccessor32<scalar_t, 2> X,
    const torch::PackedTensorAccessor32<scalar_t, 2> S,
    const torch::PackedTensorAccessor32<scalar_t, 4> C,
    const torch::PackedTensorAccessor32<scalar_t, 2> dY,
    torch::PackedTensorAccessor32<scalar_t, 2> dW,
    torch::PackedTensorAccessor32<scalar_t, 2> dS,
    torch::PackedTensorAccessor32<scalar_t, 4> dC,
    const int B, const int I, const int O, const int G)
{
    // dY [B, O]
    // dW [O, I]
    // dS [O, I]
    // dC [O, I, 2, G]

    // X [B, I]
    // S [O, I]
    // C [O, I, 2, G]

    int o = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (o < O && i < I)
    {
        scalar_t dS_sum = 0.0f;
        scalar_t dW_sum = 0.0f;
        for (int b = 0; b < B; b++)
        {
            dW_sum += dY[b][o] * X[b][i] / (1.0f + expf(-X[b][i]));
            scalar_t dS_g_sum = 0.0f;
            for (int g = 0; g < G; g++)
            {
                scalar_t v_sin, v_cos;
                sincos((g + 1) * X[b][i], &v_sin, &v_cos);
                dS_g_sum += C[o][i][0][g] * v_cos + C[o][i][1][g] * v_sin;
                dC[o][i][0][g] += dY[b][o] * S[o][i] * v_cos;
                dC[o][i][1][g] += dY[b][o] * S[o][i] * v_sin;
            }
            dS_sum += dY[b][o] * dS_g_sum;
        }
        dS[o][i] = dS_sum;
        dW[o][i] = dW_sum;
    }
}

template <typename scalar_t>
__global__ void fftkan_cuda_backward_kernel_X(
    const torch::PackedTensorAccessor32<scalar_t, 2> X,
    const torch::PackedTensorAccessor32<scalar_t, 2> W,
    const torch::PackedTensorAccessor32<scalar_t, 2> S,
    const torch::PackedTensorAccessor32<scalar_t, 4> C,
    const torch::PackedTensorAccessor32<scalar_t, 2> dY,
    torch::PackedTensorAccessor32<scalar_t, 2> dX,
    const int B, const int I, const int O, const int G)
{
    // dY [B, O]
    // dX [B, I]

    // X [B, I]
    // W [O, I]
    // S [O, I]
    // C [O, I, 2, G]

    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < B && i < I)
    {
        scalar_t sum = 0.0f;
        for (int o = 0; o < O; o++)
        {
            scalar_t g_sum = 0.0f;
            for (int g = 0; g < G; g++)
            {
                scalar_t v_sin, v_cos;
                sincos((g + 1) * X[b][i], &v_sin, &v_cos);
                g_sum -= C[o][i][0][g] * (g + 1) * v_sin;
                g_sum += C[o][i][1][g] * (g + 1) * v_cos;
            }
            sum += dY[o][i] * S[o][i] * g_sum;
            scalar_t sigmoid_x = 1 / (1 + expf(-X[b][i]));
            sum += dY[b][o] * W[o][i] * sigmoid_x * (1.0f + X[b][i] * (1 - sigmoid_x));
        }
        dX[b][i] = sum;
    }
}

torch::Tensor fftkan_cuda_forward(torch::Tensor X, torch::Tensor W, torch::Tensor S, torch::Tensor C, int B, int I, int O, int G)
{
    auto Y = torch::zeros({B, O}, X.options());

    const dim3 threads(16, 16);
    const dim3 blocks((B + threads.x - 1) / threads.x, (O + threads.y - 1) / threads.x);

    hipDeviceSynchronize(); 

    AT_DISPATCH_FLOATING_TYPES(
        X.type(),
        "fftkan_cuda_forward_kernel",
        ([&]
         { fftkan_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
               X.packed_accessor32<scalar_t, 2>(),
               W.packed_accessor32<scalar_t, 2>(),
               S.packed_accessor32<scalar_t, 2>(),
               C.packed_accessor32<scalar_t, 4>(),
               Y.packed_accessor32<scalar_t, 2>(),
               B, I, O, G); }));

    hipDeviceSynchronize();

    return Y;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor> fftkan_cuda_backward(torch::Tensor dY, torch::Tensor X, torch::Tensor W, torch::Tensor S, torch::Tensor C, int B, int I, int O, int G)
{
    auto dX = torch::zeros_like(X);
    auto dW = torch::zeros_like(W);
    auto dS = torch::zeros_like(S);
    auto dC = torch::zeros_like(C);

    const dim3 threads(16, 16);
    const dim3 blocks((O + threads.x - 1) / threads.x, (I + threads.y - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(
        X.type(),
        "fftkan_cuda_backward_WSC",
        ([&]
         { fftkan_cuda_backward_kernel_WSC<scalar_t><<<blocks, threads>>>(
               X.packed_accessor32<scalar_t, 2>(),
               S.packed_accessor32<scalar_t, 2>(),
               C.packed_accessor32<scalar_t, 4>(),
               dY.packed_accessor32<scalar_t, 2>(),
               dW.packed_accessor32<scalar_t, 2>(),
               dS.packed_accessor32<scalar_t, 2>(),
               dC.packed_accessor32<scalar_t, 4>(),
               B, I, O, G); }));


    const dim3 threads2(16, 16);
    const dim3 blocks2((B + threads2.x - 1) / threads2.x, (I + threads2.y - 1) / threads2.y);

    AT_DISPATCH_FLOATING_TYPES(
        X.type(),
        "fftkan_cuda_backward_X",
        ([&]
         { fftkan_cuda_backward_kernel_X<scalar_t><<<blocks2, threads2>>>(
               X.packed_accessor32<scalar_t, 2>(),
               W.packed_accessor32<scalar_t, 2>(),
               S.packed_accessor32<scalar_t, 2>(),
               C.packed_accessor32<scalar_t, 4>(),
               dY.packed_accessor32<scalar_t, 2>(),
               dX.packed_accessor32<scalar_t, 2>(),
               B, I, O, G); }));

    hipDeviceSynchronize();

    return {dX, dW, dS, dC};
}